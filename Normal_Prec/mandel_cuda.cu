#include "hip/hip_runtime.h"
#include "mandel_cuda.h"

void mandelbrot()
{
  int *map, *CUDA_map;
  DATA dat;
  FILE *fp;

  dat.zoom = 1;
  dat.len = 11880; dat.hei = 6480;
  dat.it = 1000, dat.colormax = 255;
  dat.cx = dat.cy = 0;

  map = (int*)malloc(sizeof(int)*dat.len*dat.hei);
  hipMalloc(&CUDA_map, sizeof(int)*dat.len*dat.hei);

  image_gen<<< (dat.len*dat.hei)/512 + 1, 512>>>(dat, CUDA_map);

  hipMemcpy(map, CUDA_map, sizeof(int)*dat.len*dat.hei, hipMemcpyDeviceToHost);

  fp = fopen("cancer.xpm", "w");
  fprintf(fp, "! XPM2\n%d %d %d 3\n", dat.len, dat.hei, dat.colormax+1);
  fprintf(fp, "000 c #000000");
  for(int i = 255; i > 0; --i){
    fprintf(fp, "\n%03d c #%02x%02x%02x", 256-i, i, i, i);
  }
  for(int i = 0; i < dat.len*dat.hei; ++i){
    if(i%dat.len == 0) fprintf(fp, "\n");
    fprintf(fp, "%03d", map[i]);
  }
  fclose(fp);
  free(map);
  hipFree(CUDA_map);
}

__global__ void image_gen(DATA dat, int* map)
{
  int N = blockDim.x*blockIdx.x + threadIdx.x;
  int x, y, i;
  double px, py, itx, ity, aux, pal;

  if(N < dat.len*dat.hei){
    x = N%dat.len;
    y = N/dat.len;

    itx = px = (l/(dat.zoom))*(((double)x)/((double)dat.len) - 0.5) + dat.cx;
    ity = py = (h/(dat.zoom))*(((double)y)/((double)dat.hei) - 0.5) + dat.cy;

    for(i = 0; i < dat.it && (itx*itx+ity*ity) < 4; ++i){
      aux = itx*itx - ity*ity + px;
      ity = 2*itx*ity + py;
      itx = aux;
    }
    if(i < dat.it){
      pal = (double)(i+1) + log2(0.5*log2(itx*itx+ity*ity));
      map[x+y*dat.len] = (int)pal%dat.colormax + 1;
    }
    else{
      map[x+y*dat.len] = 0;
    }
  }
}
