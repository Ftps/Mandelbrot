#include "hip/hip_runtime.h"
#include "h_pre.h"

int _SIZE = DEFAULT_SIZE;



void change_mant_size(int size)
{
    if(size < sizeof(int)) _SIZE = sizeof(int)*8;
    else _SIZE = size*8;
}

void check_size(NUM *n)
{
    if(n->mant_size == _SIZE) return;
    else n->mant = (bool*)realloc(n->mant, sizeof(bool)*_SIZE);

    if(_SIZE > n->mant_size){
        for(int i = n->mant_size; i < _SIZE; ++i) n->mant[i] = 0;
    }
    n->mant_size = _SIZE;

}

int is_zero(NUM *n)
{
    if(n->is_zero) return 1;

    for(int i = 0; i < _SIZE; ++i){
        if(n->mant[i]) return 0;
    }

    n->is_zero = 1;
    n->exp = n->sig = 0;
    strcpy(n->dec, ZERO);
    return 1;
}

void num_str(NUM *n)
{
    double p = 0;

    if(is_zero(n)) return;

    for(int i = 0; i < _SIZE; ++i){
        p += n->mant[i]*pow(2, n->exp - i);
    }

    (n->sig) ? sprintf(n->dec, "-%.10e", p): sprintf(n->dec, "%.10e", p);
}





NUM* init_num()
{
    NUM *new = (NUM*)malloc(sizeof(NUM));

    new->mant = (bool*)calloc(_SIZE, sizeof(bool));
    new->is_zero = 1;
    new->exp = new->sig = 0;
    new->mant_size = _SIZE;
    new->dec = (char*)malloc(sizeof(char)*(NUM_SIZE+1));
    strcpy(new->dec, ZERO);

    return new;
}

void free_num(NUM *n)
{
    free(n->mant);
    free(n->dec);
    free(n);
}

void print_num(NUM *n, bool f)
{
    num_str(n);
    (n->sig) ? printf("\n-"): putchar('\n');

    printf("%d.", n->mant[0]);
    for(int i = 1; i < _SIZE; ++i){
        (n->mant[i]) ? putchar('1'): putchar('0');
    }
    printf("e%ld\n%s\n\n", n->exp, n->dec);

    if(f) free_num(n);
}

NUM* num_cpy(NUM *n)
{
    NUM *new = init_num();

    for(int i = 0; i < _SIZE; ++i) new->mant[i] = n->mant[i];

    new->exp = n->exp;
    new->sig = n->sig;
    new->is_zero = n->is_zero;
    strcpy(new->dec, n->dec);

    return new;
}

NUM* int_num(int n)
{
    NUM *new = init_num();

    if(!n) return new;
    else if(n < 0) new->sig = 1;

    new->is_zero = 0;
    new->exp = sizeof(int)*8-1;
    n = abs(n);

    for(int i = sizeof(int)*8-1; i >= 0; --i){
        new->mant[sizeof(int)*8-1-i] = n & (0b1 << i);
    }

    while(!(new->mant[0])){
        for(int i = 0; i < sizeof(int)*8; ++i){
            new->mant[i] = new->mant[i+1];
        }
        --new->exp;
    }

    num_str(new);

    return new;
}

NUM* dou_num(double p)
{
    NUM *new = init_num();
    double n;

    if(!p) return new;
    else if(p < 0){
        new->sig = 1;
        p = abs(p);
    }
    new->is_zero = 0;
    new->exp = (int)(1+log2(p));
    p = p*pow(2, -(new->exp));

    for(int i = 0; i < _SIZE && p; ++i){
        if((n = pow(2, -i)) < p){
            new->mant[i] = 1;
            p -= n;
        }
    }

    while(!(new->mant[0])){
        for(int i = 0; i < _SIZE-1; ++i){
            new->mant[i] = new->mant[i+1];
        }
        --new->exp;
    }

    num_str(new);

    return new;

}






int compare_num(NUM *n1, NUM *n2)
{
    if(n1->exp > n2->exp) return 0b01;
    else if(n2->exp > n1->exp) return 0b10;
    else{
        for(int i = 0; i < _SIZE; ++i){
            if(n1->mant[i] != n2->mant[i]){
                if(n1->mant[i]) return 0b01;
                else return 0b10;
            }
        }
    }

    return 0b00;
}

NUM* add_num(NUM *n1, NUM *n2, int f)
{
    NUM *add, *big, *small;
    int cmp, a = 1, carry = 0, exp_dif, i;

    if(is_zero(n1)){add = num_cpy(n2); a = 0;}
    else if(is_zero(n2)){add = num_cpy(n1); a = 0;}
    else if((cmp = compare_num(n1, n2)) & 0b01){big = n1; small = n2;}
    else if(cmp & 0b10){big = n2; small = n1;}
    else{
        a = 0;
        if(n1->sig != n2->sig) add = init_num();
        else{
            add = num_cpy(n1);
            ++add->exp;
        }
    }

    if(a){
        if((exp_dif = big->exp - small->exp) >= _SIZE) add = num_cpy(big);
        else{
            a = (big->sig == small->sig) ? 0b0: 0b1;
            add = init_num();
            add->is_zero = 0;
            add->exp = big->exp;
            add->sig = big->sig;

            for(i = _SIZE-1; i >= exp_dif; --i){
                add->mant[i] = (big->mant[i]^(small->mant[i-exp_dif]^a))^carry;
                if((big->mant[i]+(small->mant[i-exp_dif]^a)+carry) > 1) carry = 1;
                else carry = 0;
            }
            for(int k = i; k >= 0; --k){
                add->mant[k] = (big->mant[k]^carry)^a;
                if((big->mant[k]+carry+a) > 1) carry = 1;
                else carry = 0;
            }
            if(carry && !a){
                for(int k = _SIZE-2; k >= 0; --k){
                    add->mant[k+1] = add->mant[k];
                }
                add->mant[0] = 1;
                ++add->exp;
            }
        }
    }

    if(f & 0b01) free_num(n1);
    if(f & 0b10) free_num(n2);

    return add;
}

NUM* mult_num(NUM *n1, NUM *n2, int f)
{
    NUM *mult = init_num(), *extra;

    if(n1->is_zero || n2->is_zero) return mult;

    extra = num_cpy(n1);
    extra->sig = 0;

    for(int i = 0; i < _SIZE; ++i){
        if(n2->mant[_SIZE-1-i]){
            extra->exp = i;
            mult = add_num(mult, extra, 0b01);
        }
    }

    if(n1->sig != n2->sig) mult->sig = 1;
    mult->exp += n1->exp + n2->exp - _SIZE+1;

    free_num(extra);

    if(f & 0b01) free_num(n1);
    if(f & 0b10) free_num(n2);

    return mult;
}

NUM* inv_num(NUM* n, bool f)
{
    NUM *iter, *two, *aux;
    double est;

    if(n->is_zero) PRINT_ERROR("ERR: DIVIDE BY ZERO", ERR_DIV_ZERO)

    num_str(n);
    sscanf(n->dec, "%lf", &est);
    iter = dou_num(1/est);
    two = int_num(2);

    for(int i = 0; i < (log2(_SIZE)+1); ++i){
        aux = mult_num(iter, n, 0b00);
        aux->sig = aux->sig^0b1;
        iter = mult_num(iter, add_num(two, aux, 0b10), 0b11);
    }

    free_num(two);
    if(f) free_num(n);

    return iter;
}

NUM* div_num(NUM *n1, NUM *n2, int f)
{
    NUM *inv, *div;

    if(n1->is_zero && !n1->is_zero) div = init_num();
    else{
        inv = inv_num(n2, 0b00);
        div = mult_num(n1, inv, 0b10);
    }

    if(f & 0b01) free_num(n1);
    if(f & 0b10) free_num(n2);

    return div;
}
