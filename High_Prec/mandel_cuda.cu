#include "hip/hip_runtime.h"
#include "mandel_cuda.h"

// FOR TEST NOW
void mandelbrot()
{
  int *map, *CUDA_map;
  DATA dat;

  dat.pre = 1;
  dat.cx = generate(dat.pre);
  dat.cy = generate(dat.pre);
  dat.len = 1980; dat.hei = 1080;
  dat.zoom = 1;
  dat.it = 100;
  dat.colormax = 1;

  dat.CUDA_cx = num2CUDA(dat.cx, dat.pre);
  dat.CUDA_cy = num2CUDA(dat.cy, dat.pre);

  map = (int*)malloc(sizeof(int)*dat.len*dat.hei);
  hipMalloc(&CUDA_map, sizeof(int)*dat.len*dat.hei);

  image_gen<<< (dat.len*dat.hei+SIZE_THREAD-1)/SIZE_THREAD, SIZE_THREAD>>>(dat, CUDA_map);

  hipMemcpy(map, CUDA_map, sizeof(int)*dat.len*dat.hei, hipMemcpyDeviceToHost);

  for(int i = 0; i < dat.len*dat.hei; ++i){
    if(i%dat.len == 0) putchar('\n');
    if(map[i] == 0) printf("* ");
    else printf(". ");
  }

  putchar('\n');

  hipFree(CUDA_map);
  free(map);
}







int test(int argc, char* argv[])
{
  int pre = 4;

  if(argc == 3 && !strcmp("-j", argv[1])){
    sscanf(argv[2], "%d", &pre);
    if(pre%deci == 0) pre = pre/deci;
    else pre = pre/deci + 1;
  }

  return pre;
}

void printnum(num *n1, int pre, int free)
{
  //long double c = (10.0*(long double)n1->mant[0])/((long double)powdeci);

  if(n1->signal == 0){
    printf("0.000e0\n");
    if(free) free_num(n1);
    return;
  }
  if(n1->signal == -1) printf("-");
  //printf("%.16llf", c);*/
  for(int i = 0; i < pre; ++i){
    printf("%017ld", n1->mant[i]);
  }
  printf("e%d\n\n", n1->exp);
  if(free) free_num(n1);
}

num generaterand(int pre)
{
  num gen;
  gen.exp = rand()%12 + 1;
  do{
    gen.signal = ((rand()%3)-1);
    printf("Signal = %d\n", gen.signal);
  }while(gen.signal == 0);

  for(int i = 0; i < pre; ++i){
    gen.mant[i] = (((long int)rand())*((long int)rand()))%powdeci;
  }

  return gen;
}

double rand_double()
{
  double p;

  p = (double)((rand()*rand())%powdeci)*(double)pow(10, (rand()%24)-11);

  return p;
}

__device__ double mod_numAp(num *n1, num *n2)
{
  double p = 0, n11, n12;

  n11 = (((double)10*n1->mant[0])/powdeci)*(((double)10*n1->mant[0])/powdeci);

  n12 = (((double)10*n2->mant[0])/powdeci)*(((double)10*n2->mant[0])/powdeci);

  if(n1->exp > -4) p += n11*pow(10, 2*n1->exp);
  if(n2->exp > -4) p += n12*pow(10, 2*n2->exp);

  return p;
}





__global__ void image_gen(DATA dat, int* map)
{
  num *posx, *posy, *intx, *inty, *aux;
  int x, y;
  int N = blockDim.x*blockIdx.x + threadIdx.x;
  int i;
  double pal;

  if(N < dat.len*dat.hei){
    x = N%dat.len;
    y = N/dat.len;

    //posx = d2n((l/(dat.zoom))*(((double)x)/((double)dat.len) - 0.5), dat.pre);
    //posy = d2n((h/(dat.zoom))*(((double)y)/((double)dat.hei) - 0.5), dat.pre);

    posx = generateCUDA(dat.pre);
    posy = generateCUDA(dat.pre);

    posx = addnumber(posx, dat.CUDA_cx, dat.pre, 0, 1);
    posy = addnumber(posy, dat.CUDA_cy, dat.pre, 0, 1);

    intx = num_cpy(posx, dat.pre);
    inty = num_cpy(posy, dat.pre);

    for(i = 0; (i < dat.it) && (pal = mod_numAp(intx, inty) < 4); ++i){
      aux = addnumber(sq_num(intx, dat.pre, 0, 0), sq_num(inty, dat.pre, 1, 0), dat.pre, 0, 3);
      aux = addnumber(aux, posx, dat.pre, 0, 1);
      inty = opt_mult(2, multiply(intx, inty, dat.pre, 3), dat.pre, 1, 1);
      inty = addnumber(inty, posy, dat.pre, 0, 1);
      intx = aux;
    }

    free_numCUDA(posx);
    free_numCUDA(posy);
    free_numCUDA(intx);
    free_numCUDA(inty);

    /*if(i < dat.it){
      pal = (double)(i+1) + log2(0.5*log2(pal));
      map[x+y*dat.len] = ((int)pal)%dat.colormax + 1;
      //map[x+y*dat.len] = 1;
    }
    else{*/
      map[x+y*dat.len] = 0;
    //}
  }
}




__global__ void test_func(double* line, int size, int pre)
{
  int N = blockIdx.x*blockDim.x + threadIdx.x;
  num *aux;

  if(N < size){
    aux = d2n(line[N], pre);
    aux = addnumber(aux, aux, pre, 0, 1);
    line[N] = n2d(aux, 1);
  }
}

void test(){
  double *line, *CUDA_line;

  line = (double*)malloc(sizeof(double)*40);
  gpuErrchk(hipMalloc(&CUDA_line, sizeof(double)*40));
  for(int i = 0; i < 40; ++i){
    line[i] = rand_double();
    printf("%lf ", line[i]);
  }
  gpuErrchk(hipMemcpy(CUDA_line, line, sizeof(double)*40, hipMemcpyHostToDevice));

  test_func<<<40/SIZE_THREAD + 1, SIZE_THREAD>>>(CUDA_line, 40, 2);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  gpuErrchk(hipMemcpy(line, CUDA_line, sizeof(double)*40, hipMemcpyDeviceToHost));

  printf("\n\n\n\n");
  for(int i = 0; i < 40; ++i){
    printf("%lf ", line[i]);
  }
}

void test2()
{
  double *lin1, *lin2;
  num *aux;
  int pre = 2, size = 2000;
  lin1 = (double*)malloc(sizeof(double)*size);
  lin2 = (double*)malloc(sizeof(double)*size);

  for(int i = 0; i < size; ++i){
    lin1[i] = rand_double();
  }

  for(int i = 0; i < size; ++i){
    aux = d2n(lin1[i], pre);
    aux = addnumber(aux, sq_num(aux, pre, 1, 0), pre, 0, 3);
    lin2[i] = n2d(aux, 1);
    printf("%lf - %lf^2 = %lf\n", lin1[i], lin1[i], lin2[i]);
  }
  free(lin1);
  free(lin2);
}
